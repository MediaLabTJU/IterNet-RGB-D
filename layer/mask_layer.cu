#include <vector>
#include "caffe/mask_layer.hpp"

namespace caffe {

template <typename Dtype>
void MaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
	Dtype* top_data = top[0]->mutable_gpu_data();
	switch (this->layer_param_.mask_param().masktype()) {
	case MaskParameter_MaskType_VERTICAL:
		if (this->layer_param_.mask_param().direction() 
				== MaskParameter_Direction_TOP2BOTTOM)
		{
			caffe_gpu_set(width_, (Dtype)0, top_data);
			top_data += top[0]->offset(1);
			caffe_gpu_set((height_ - 1)* width_, (Dtype)1, top_data);
		}
		else if (this->layer_param_.mask_param().direction() 
			== MaskParameter_Direction_BOTTOM2TOP)
		{
			caffe_gpu_set((height_ - 1)* width_, (Dtype)1, top_data);
			top_data += top[0]->offset(1);
			caffe_gpu_set(width_, (Dtype)0, top_data);
		}
		break;
	case MaskParameter_MaskType_HORIZONTAL:
		if (this->layer_param_.mask_param().direction()
				== MaskParameter_Direction_LEFT2RIGHT)
		{
			for (int i = 0; i < height_; i++)
			{
				caffe_gpu_set(1, (Dtype)0, top_data);
				caffe_gpu_set(width_ - 1, (Dtype)1, top_data + 1);
				top_data += top[0]->offset(1);
			}
		}
		else if (this->layer_param_.mask_param().direction()
					 == MaskParameter_Direction_RIGHT2LEFT)
		{
			for (int i = 0; i < height_; i++)
			{	
				caffe_gpu_set(width_ - 1, (Dtype)1, top_data + 1);
				caffe_gpu_set(1, (Dtype)0, top_data + width_ - 1);
				top_data += top[0]->offset(1);
			}			
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(MaskLayer);
}
