#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/transpose_layer.hpp"

namespace caffe {

template <typename Dtype>
void TransposeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  	const Dtype* bottom_data = bottom[0]->gpu_data();
  	Dtype* top_data = top[0]->mutable_gpu_data();
  	switch (this->layer_param_.transpose_param().transposetype()) {
    case TransposeParameter_TransposeType_CONVTOLSTM:
		for (int n = 0; n < bottom[0]->num(); ++n) 
		{
			const int spatial_dim = bottom[0]->width() * bottom[0]->height();
			caffe_gpu_transpose(bottom[0]->channels(), spatial_dim,
			    bottom_data + bottom[0]->offset(n),
			    top_data + top[0]->offset(n * spatial_dim));
		}
	break;
	case TransposeParameter_TransposeType_LSTMTOCONV:
		for (int n = 0; n < bottom[0]->num(); ++n) 
		{
			const int spatial_dim = bottom[0]->width() * bottom[0]->height();
			caffe_gpu_transpose(bottom[0]->width() * bottom[0]->height(),
			    bottom[0]->channels(), bottom_data + bottom[0]->offset(n),
			    top_data + top[0]->offset(n * spatial_dim));
		}	
	break;
	}	
}

template <typename Dtype>
void TransposeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  	/*Backward_cpu(top, propagate_down, bottom);*/
  	const Dtype* top_diff = top[0]->gpu_diff();
  	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  	switch (this->layer_param_.transpose_param().transposetype()) {
    case TransposeParameter_TransposeType_CONVTOLSTM:
		for (int n = 0; n < bottom[0]->num(); ++n) 
		{
			const int spatial_dim = bottom[0]->width() * bottom[0]->height();
			caffe_gpu_transpose(bottom[0]->width() * bottom[0]->height(),
			    bottom[0]->channels(), top_diff + top[0]->offset(n * spatial_dim),
			    bottom_diff + bottom[0]->offset(n));
		}
	break;
	case TransposeParameter_TransposeType_LSTMTOCONV:
		for (int n = 0; n < bottom[0]->num(); ++n) 
		{
			const int spatial_dim = bottom[0]->width() * bottom[0]->height();
			caffe_gpu_transpose(bottom[0]->channels(), spatial_dim,			    
			    top_diff + top[0]->offset(n * spatial_dim),
			    bottom_diff + bottom[0]->offset(n));
		}
	break;
	}		

}


INSTANTIATE_LAYER_GPU_FUNCS(TransposeLayer);

} // namespace caffe
